#include "hip/hip_runtime.h"
#include "rng.cuh"
#include <iostream>
#include <cmath>
#include <cstdlib>

__device__ __host__ rng::rng(){
  m_s1 = 0;
  m_s2 = 0;
  m_s3 = 0;
  m_s4 = 0;

  m_gauss1 = 0;
  m_gauss2 = 0;
}

__device__ __host__ rng::rng(unsigned s1, unsigned s2, unsigned s3, unsigned s4) {

  m_s1 = s1;
  m_s2 = s2;
  m_s3 = s3;
  m_s4 = s4;

  m_gauss1 = 0;
  m_gauss2 = 0;

}

__device__ __host__ rng::rng(const rng &rng2){

  m_s1 = rng2.m_s1;
  m_s2 = rng2.m_s2;
  m_s3 = rng2.m_s3;
  m_s4 = rng2.m_s4;

  m_gauss1 = rng2.m_gauss1;
  m_gauss2 = rng2.m_gauss2;

}

__device__ __host__ rng::~rng() {}

__device__ __host__ unsigned rng::TauStep (unsigned &seed, int k1, int k2, int k3, unsigned M) {

  unsigned b = (((seed << k1)^seed)>>k2);
  seed = ((seed&M)<<k3)^b;
  return seed;

}

__device__ __host__ unsigned rng::LNG (unsigned &seed, unsigned a, unsigned b) {

  return seed = (a*seed + b);

}

__device__ __host__ double rng::hybrid() /*(unsigned s1, unsigned s2, unsigned s3, unsigned s4) */{

  return (2.3283064365387e-10*(TauStep(m_s1, 13, 19, 12, 4294967294UL) ^
  (TauStep(m_s2, 2, 25, 4, 4294967288UL) )^
  (TauStep(m_s3, 3, 11, 17, 4294967280UL))
  ^(LNG(m_s4, 1664525, 1013904223UL))));

}
__device__ __host__ double rng::Get_uniform() {

  return rng::hybrid();
}

__device__ __host__ double rng::Get_gauss() {

    double u = sqrt(-2*log(rng::Get_uniform()))*cos(2*M_PI*rng::Get_uniform());
    double v = sqrt(-2*log(rng::Get_uniform()))*sin(2*M_PI*rng::Get_uniform());

    return u;
};
