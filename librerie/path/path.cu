#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <cstdlib>
#include "path.h"

__device__ __host__ path::path() {

  m_volatility      = 0;
  m_risk_free_rate  = 0;
  m_prezzo_iniziale = 0;

  m_price           = 0;

  m_option_type     = 0;
  m_numero_steps    = 0;
  m_delta_time      = 0;
  m_time_to_maturity= 0;
  m_strike_price    = 0;

  m_payoff = 0;

  m_B = 0;
  m_K = 0;
  m_N = 0;
  m_corridor = 0;

}



__device__ __host__ path::path(input_option_data option, input_market_data market) {

  m_volatility      = (market.volatility);
  m_risk_free_rate  = (market.risk_free_rate);
  m_prezzo_iniziale = (market.prezzo_iniziale);

  m_price           = market.prezzo_iniziale;

  m_option_type     = (option.option_type);
  m_numero_steps    = (option.numero_steps);
  m_delta_time      = (option.delta_time);
  m_time_to_maturity= (option.time_to_maturity);
  m_strike_price    = (option.strike_price);

  m_payoff = 0;

  m_B = (option.B);
  m_K = (option.K);
  m_N = (option.N);

  m_corridor = 0;
  // m_date_fixing = date_fixing;
}

// copy costructor

/*__device__ __host__ path::path(const path &path2) {
  m_p            = path2.m_p;
  volatility          = path2.volatility;
  risk_free_rate = path2.risk_free_rate;

  m_strike_price            = path2.m_strike_price;
  dt             = path2.dt;
  m_n_steps      = path2.m_n_steps;

}*/



//destructor
__device__ __host__ path::~path() {

  } ;

__device__ __host__ double path::GetPrice() {
  return m_price;
}

__device__ __host__ void path::corridor_controller(double prezzo1, double prezzo2) {

  double value_to_be_checked = fabs((1/sqrt(m_delta_time))*log(prezzo2/prezzo1));
  double barriera            = m_B * m_volatility;

    if (value_to_be_checked < barriera) {

      m_corridor ++ ;

    }

}

__device__ __host__ void path::eulero(double gauss) {

    double price_i;
    price_i = m_price * (1 + m_risk_free_rate*m_delta_time + m_volatility*sqrt(m_delta_time)*gauss);

      if (m_option_type == 2) {
        corridor_controller(price_i, m_price);
      }

    m_price = price_i;

    // check negative price
}





  //payoff

__device__ __host__ void path::payoff_evaluator() {

  //  OPTION TIPES: f(orward) - p(lain vanilla) - c(orridor)

  if (m_option_type == 0) {

    m_payoff = m_price;

  }

  if (m_option_type == 1) {
    // double payoff = 0;
    m_payoff = max(m_price - m_strike_price, 0.);
    // return payoff;
  }

  if (m_option_type == 2) {

    m_payoff = m_N *(max((1/m_numero_steps)*m_corridor - m_K, 0.));

  }
  // else m_payoff = -1000;

}





__device__ __host__ double path::Get_payoff() {

  return m_payoff;

}
