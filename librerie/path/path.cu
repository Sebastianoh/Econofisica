#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <cstdlib>
#include "path.h"

__device__ __host__ path::path() {

  m_volatility      = 0;
  m_risk_free_rate  = 0;
  m_prezzo_iniziale = 0;

  m_price           = 0;

  m_option_type     = 0;
  m_numero_steps    = 0;
  m_delta_time      = 0;
  m_time_to_maturity= 0;
  m_strike_price    = 0;

  m_payoff = 0;

  m_B = 0;
  m_K = 0;
  m_N = 0;
  m_corridor = 0;
  m_test_variable = 0;

}



__device__ __host__ path::path(input_option_data option, input_market_data market) {

  m_volatility      = (market.volatility);
  m_risk_free_rate  = (market.risk_free_rate);
  m_prezzo_iniziale = (market.prezzo_iniziale);

  m_price           = market.prezzo_iniziale;

  m_option_type     = (option.option_type);
  m_numero_steps    = (option.numero_steps);
  m_delta_time      = (option.delta_time);
  m_time_to_maturity= (option.time_to_maturity);
  m_strike_price    = (option.strike_price);

  m_payoff = 0;

  m_B = (option.B);
  m_K = (option.K);
  m_N = (option.N);

  m_corridor = 0;


  m_test_variable = 0;
}

//destructor
__device__ __host__ path::~path() {

}

__device__ __host__ double path::GetPrice() {
  return m_price;
}


__device__ __host__ double path::Get_controller() {
    return m_corridor;
}



__device__ __host__ void path::corridor_controller(double prezzo1, double prezzo2) {

  double value_to_be_checked = (1./sqrt(m_delta_time)) *log( prezzo2/ prezzo1 );
  double barriera            = m_B * m_volatility;

    if ( fabs(value_to_be_checked) < barriera) {
      ++(m_corridor);
    }

}

__device__ __host__ void path::eulero(double gauss) {

  double price_i;
  price_i = m_price * (1 + m_risk_free_rate*m_delta_time + m_volatility*sqrt(m_delta_time)*gauss);

    if (m_option_type == 2) {
      corridor_controller(m_price, price_i);
      m_test_variable ++;
    }

  m_price = price_i;

    // check negative price

}

__device__ __host__ void  path::exact(double gauss) {

  price_i = m_price * exp(m_risk_free_rate -0.5*pow(m_volatility,2)*(m_delta_time)
          + m_volatility*gauss*sqrt(m_delta_time));

    if (m_option_type == 2) {
      corridor_controller(m_price, price_i);
      m_test_variable ++;
    }

  m_price = price_i;

}

  //payoff

__device__ __host__ void path::payoff_evaluator() {

  //  OPTION TIPES: f(orward) - p(lain vanilla) - c(orridor)

  if (m_option_type == 0) {

    m_payoff = m_price;

  }

  if (m_option_type == 1) {
    // double payoff = 0;
    m_payoff = max(m_price - m_strike_price, 0.);
    // return payoff;
  }

  if (m_option_type == 2) {

    m_payoff = m_N *(max(( (1./m_numero_steps) * m_corridor ) - m_K, 0.));

  }
  // else m_payoff = -1000;

}





__device__ __host__ double path::Get_payoff() {

  return m_payoff;

}
