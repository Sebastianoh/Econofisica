#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <cstdlib>
#include <algorithm>
#include "statistica.h"

__device__ __host__ statistica::statistica(input_mc_data M) {

   m_somma_payoff        = 0;
   m_somma_quadra_payoff = 0;
   m_deviazione_standard = 0;
   m_num_simulazioni     = M.N_simulazioni;
}

__device__ __host__ void statistica::analisi(double payoff) {

  m_somma_payoff        += payoff;
  m_somma_quadra_payoff += pow(payoff,2);
  m_deviazione_standard += pow(payoff - m_media, 2);
}


__device__ __host__ double statistica::get_somma_payoff() {
  return m_somma_payoff;
}

__device__ __host__ double statistica::get_somma_quadra_payoff() {
  return m_somma_quadra_payoff;
}


__device__ __host__ double statistica::get_media() {
  return m_somma_payoff/m_num_simulazioni;
}

__device__ __host__ double statistica::get_deviazione_standard() {
  return sqrt(m_deviazione_standard/m_num_simulazioni);
}
