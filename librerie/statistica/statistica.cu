#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <cstdlib>
#include <algorithm>
#include "statistica.h"


//si è reso necessario introdurre variabile n_step_simulazione poiché
// usando il numero fixed di simulazioni quando avevo 1 solo numero,
// la media e dev_std venivano calcolate comunque con n = n_sim massimo


// a questo punto il numero fissato di simulazioni diventa una variabile
//  inutile quindi la cancello

__device__ __host__ statistica::statistica() {

  m_somma_payoff        = 0;
  m_somma_quadra_payoff = 0;
  m_deviazione_standard = 0;
  m_step_simulazione    = 0;
  // m_num_simulazioni     = 1;

}


__device__ __host__ void statistica::analisi(double payoff) {

  ++ m_step_simulazione ;

  m_somma_payoff        += payoff;
  m_somma_quadra_payoff += pow(payoff,2);
  m_deviazione_standard += pow(payoff - (m_somma_payoff/m_step_simulazione), 2);

}


__device__ __host__ double statistica::get_somma_payoff() {
  return m_somma_payoff;
}

__device__ __host__ double statistica::get_somma_quadra_payoff() {
  return m_somma_quadra_payoff;
}


__device__ __host__ double statistica::get_media() {
  return m_somma_payoff/m_step_simulazione;
}

__device__ __host__ double statistica::get_deviazione_standard() {
  return sqrt(m_deviazione_standard/m_step_simulazione);
}
