#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <cstdlib>
#include <algorithm>
#include "statistica.h"

__device__ __host__ statistica::statistica(input_mc_data M) {

   m_size = M.N_simulazioni;  //la size del vettore è il numero di simulazioni montecarlo
    //passo la struttura con dentro anche il numero di simulazioni monte a questa classe così lo conosco
    // for (size_t i = 0; i < m_size; i++) {
      // m_array[i] = array [i];
    // }

};

__device__ __host__ double media(double * array) {
  double somma = 0;
  double media = 0;
    for (size_t i = 0; i < m_size; i++) {
      double somma_dummy = array[i];
      somma += somma_dummy;
    }
  media = somma/m_size;
  return media;
}
