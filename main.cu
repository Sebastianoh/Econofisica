#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <cstdlib>
#include "funzioni.h"
#include "rng.h"
#include "rng.cu"
#include "processo_stocastico.h"
#include "processo_stocastico.cu"

using namespace std;
//le global non posso definirle nelle classi

__global__ void pricer_montecarlo(processo_stocastico* pricer ,double * array_prezzi_finali) {

    int i = threadIdx.x + blockDim.x*blockIdx.x;
    // uso -> perché gli passo una classe by pointer
    array_prezzi_finali[i] = pricer->Get_new_price();

};

int main() {

  int N = 15000;

  unsigned s1 = 0;
  unsigned s2 = 0;
  unsigned s3 = 0;
  unsigned s4 = 0;

  random_seed_generator(s1,s2,s3,s4);

  rng random_number_generator(s1,s2,s3,s4);
  processo_stocastico host_pricer(10, 50, 12);

  double * prezzi     = new double [N];
  double * dev_prezzi = new double [N];

  //storage su cui copiare host pricer

  processo_stocastico *dev_pricer;

  hipMalloc( (void **)&dev_pricer, N*sizeof(processo_stocastico) );
  hipMalloc( (void **)&dev_prezzi, N*sizeof(double) );

  // copio:
  hipMemcpy(&dev_pricer, &host_pricer, N*sizeof(processo_stocastico), hipMemcpyHostToDevice);

  //prova senza troppe pretese,
  // still non funziona

  pricer_montecarlo<<<10,2>>>(dev_pricer, dev_prezzi);

  for (size_t i = 0; i < 15; i++) {
    std::cout << "prezzi : " << dev_prezzi[i] << '\n';
  };

  hipMemcpy(prezzi, dev_prezzi, N*sizeof(double), hipMemcpyDeviceToHost);

  hipFree(dev_pricer);
  hipFree(dev_prezzi);

    // prova
    // for (size_t i = 0; i < 15; i++) {
      // std::cout << "prezzi: " << prezzi[i] << '\n';
    // };

















return 0;
}
