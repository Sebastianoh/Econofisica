#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <cstdlib>
#include "path.h"
#include "rng.h"

__device__ __host__ path::path() {
  m_p            = 0;
  m_E            = 0;
  dt             = 0.5;   //valore a caso
  sigma          = 0.5;   //valore a caso
  risk_free_rate = 0.02;
  m_n_steps      = 0;
}



__device__ __host__ path::path(input_option_data option, input_market_data market) {
  m_p            = market.S_0;
  sigma          = market.sigma;
  risk_free_rate = market.risk_free_rate;

  m_E            = option.E;
  dt             = option.delta_t;
  m_n_steps      = option.num_intervals;

}

__device__ __host__ path::path(const path &path2) {
  m_p            = path2.m_p;
  sigma          = path2.sigma;
  risk_free_rate = path2.risk_free_rate;

  m_E            = path2.m_E;
  dt             = path2.dt;
  m_n_steps      = path2.m_n_steps;

}



//destructor
__device__ __host__ path::~path() {
}

__device__ __host__ double path::eulero(double gauss) {
    double S;
    S = m_p;
    S = S*(1 + risk_free_rate*dt + sigma*sqrt(dt)*gauss);
  return S;
}


/*__device__ __host__ double path::Get_new_price() {
  creazione_path();
  double dummy = m_p;
  return dummy;

}
  //payoff

__device__ __host__ double path::payoff() {

  return max((m_p - m_E), 0.);

}
*/
