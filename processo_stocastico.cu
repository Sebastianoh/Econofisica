#include "hip/hip_runtime.h"
#include "processo_stocastico.h"
#include <iostream>
#include <cmath>
#include <cstdlib>

processo_stocastico::processo_stocastico(double p, int n_steps) {

  m_p            = p;
  dt             = 0.5;   //valore a caso
  sigma          = 0.5;   //valore a caso
  risk_free_rate = 0.02;
  m_n_steps = n_steps;
}
//destructor
processo_stocastico::~processo_stocastico() {
}

void processo_stocastico::eulero(double& S) {

  S = S*(1 + risk_free_rate*dt + sigma*sqrt(dt)*rng::Get_gauss());

}

void processo_stocastico::creazione_path(){

  for (size_t i = 0; i < m_n_steps; i++) {
    eulero(m_p);
    // std::cout << "p ad ogni step: " <<  m_p <<'\n';          CAMBIANO CASUALMENTE POI DIVENTANO MINORI DI 1 MA PENSO SIA
    //                                                          PERCHE' HO MESSO NUMERI A CASO
  };
}

double processo_stocastico::Get_new_price() {
  creazione_path();
  return m_p;
}
