#include "hip/hip_runtime.h"
#include "processo_stocastico.h"
#include <iostream>
#include <cmath>
#include <cstdlib>

processo_stocastico::processo_stocastico(double p) {

  m_p            = p;
  dt             = 0.5;   //valore a caso
  sigma          = 0.5;   //valore a caso
  risk_free_rate = 0.02;

}
//destructor
processo_stocastico::~processo_stocastico() {
}

void processo_stocastico::eulero(double& S) {

  S = S*(1 + risk_free_rate*dt + sigma*sqrt(dt)*rng::Get_gauss());

}

double processo_stocastico::Get_new_price() {
  eulero(m_p);
  return m_p;
}
