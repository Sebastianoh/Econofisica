#include "hip/hip_runtime.h"
#include "processo_stocastico.h"
#include <iostream>
#include <cmath>
#include <cstdlib>

processo_stocastico::processo_stocastico(double p, int n_steps, double E) {

  m_p            = p;
  m_E            = E;
  dt             = 0.5;   //valore a caso
  sigma          = 0.5;   //valore a caso
  risk_free_rate = 0.02;
  m_n_steps      = n_steps;
}
//destructor
processo_stocastico::~processo_stocastico() {
}

void processo_stocastico::eulero(double& S) {

  S = S*(1 + risk_free_rate*dt + sigma*sqrt(dt)*rng::Get_gauss());
}

void processo_stocastico::creazione_path(){

  for (size_t i = 0; i < m_n_steps; i++) {
    eulero(m_p);
  };
}

double processo_stocastico::Get_new_price() {
  creazione_path();
  return m_p;
}

double processo_stocastico::payoff() {

  return max((m_p - m_E), 0);

}
