#include "hip/hip_runtime.h"
#include "processo_stocastico.h"
#include <iostream>
#include <cmath>
#include <cstdlib>

__device__ __host__ processo_stocastico::processo_stocastico() {
  m_p            = 0;
  m_E            = 0;
  dt             = 0.5;   //valore a caso
  sigma          = 0.5;   //valore a caso
  risk_free_rate = 0.02;
  m_n_steps      = 0;
}



__device__ __host__ processo_stocastico::processo_stocastico(input_option_data option, input_market_data market) {
  m_p            = market.S_0;
  sigma          = market.sigma;
  risk_free_rate = market.risk_free_rate;

  m_E            = option.E;
  dt             = option.delta_t;
  m_n_steps      = option.num_intervals;
}
//destructor
__device__ __host__ processo_stocastico::~processo_stocastico() {
}

__device__ __host__ void processo_stocastico::eulero(double& S) {

  S = S*(1 + risk_free_rate*dt + sigma*sqrt(dt)*rng::Get_gauss());
}

__device__ __host__ void processo_stocastico::creazione_path(){

  for (size_t i = 0; i < m_n_steps; i++) {
    eulero(m_p);
  };
}

__device__ __host__ double processo_stocastico::Get_new_price() {
  creazione_path();
  return m_p;
}

  //payoff

__device__ __host__ double processo_stocastico::payoff() {

  return max((m_p - m_E), 0.);

}
