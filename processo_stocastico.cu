#include "hip/hip_runtime.h"
#include "processo_stocastico.h"
#include <iostream>
#include <cmath>
#include <cstdlib>

processo_stocastico::processo_stocastico() {
  m_p            = 0;
  m_E            = 0;
  dt             = 0.5;   //valore a caso
  sigma          = 0.5;   //valore a caso
  risk_free_rate = 0.02;
  m_n_steps      = 0;
}



processo_stocastico::processo_stocastico(double p, int n_steps, double E) {

  m_p            = p;
  m_E            = E;
  dt             = 0.5;   //valore a caso
  sigma          = 0.5;   //valore a caso
  risk_free_rate = 0.02;
  m_n_steps      = n_steps;
}
//destructor
processo_stocastico::~processo_stocastico() {
}

__device__ __host__ void processo_stocastico::eulero(double& S) {

  S = S*(1 + risk_free_rate*dt + sigma*sqrt(dt)*rng::Get_gauss());
}

__device__ __host__ void processo_stocastico::creazione_path(){

  for (size_t i = 0; i < m_n_steps; i++) {
    eulero(m_p);
  };
}

__device__ __host__ double processo_stocastico::Get_new_price() {
  creazione_path();
  return m_p;
}

  //payoff

__device__ __host__ double processo_stocastico::payoff() {

  return max((m_p - m_E), 0.);

}
