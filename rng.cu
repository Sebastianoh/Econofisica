#include "hip/hip_runtime.h"
#include "rng.h"
#include <iostream>
#include <cmath>
#include <cstdlib>

rng::rng(){
  m_s1 = 0;
  m_s2 = 0;
  m_s3 = 0;
  m_s4 = 0;

  m_gauss1 = 0;
  m_gauss2 = 0;
}

rng::rng(unsigned s1, unsigned s2, unsigned s3, unsigned s4) {

  m_s1 = s1;
  m_s2 = s2;
  m_s3 = s3;
  m_s4 = s4;

}

rng::~rng() {}

__device__ __host__ unsigned rng::TauStep (unsigned &seed, int k1, int k2, int k3, unsigned M) {

  unsigned b = (((seed << k1)^seed)>>k2);
  seed = ((seed&M)<<k3)^b;
  return seed;

}

__device__ __host__ unsigned rng::LNG (unsigned &seed, unsigned a, unsigned b) {

  return seed = (a*seed + b);

}

__device__ __host__ double rng::hybrid() /*(unsigned s1, unsigned s2, unsigned s3, unsigned s4) */{

  return (2.3283064365387e-10*(TauStep(m_s1, 13, 19, 12, 4294967294UL) ^
  (TauStep(m_s2, 2, 25, 4, 4294967288UL) )^
  (TauStep(m_s3, 3, 11, 17, 4294967280UL))
  ^(LNG(m_s4, 1664525, 1013904223UL))));

}
__device__ __host__ double rng::Get_uniform() {

  return rng::hybrid();
}


__device__ __host__ void rng::BoxMuller (double &u, double &v) {

    u = sqrt(-2*log(rng::Get_uniform()))*cos(2*M_PI*rng::Get_uniform());
    v = sqrt(-2*log(rng::Get_uniform()))*sin(2*M_PI*rng::Get_uniform());

}

__device__ __host__ double rng::Get_gauss() {
  rng::BoxMuller(m_gauss1, m_gauss2);
  return m_gauss1;
};
