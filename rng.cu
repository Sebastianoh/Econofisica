#include "hip/hip_runtime.h"
#include "rng.h"
#include <iostream>
#include <cmath>
#include <cstdlib>

rng::rng(){}

rng::rng(unsigned s1, unsigned s2, unsigned s3, unsigned s4) {
  m_s1 = s1;
  m_s2 = s2;
  m_s3 = s3;
  m_s4 = s4;

}

rng::~rng() {}

/*void rng::first_generator() {

  srand(time(NULL));
  m_s1 = rand() % 1000000+128;
  m_s2 = rand() % 1000000+128;
  m_s3 = rand() % 1000000+128;
  m_s4 = rand() % 1000000+128;

}*/

__device__ unsigned rng::TauStep (unsigned seed, int k1, int k2, int k3, unsigned M) {

  unsigned b = (((seed << k1)^seed)>>k2);
  seed = ((seed&M)<<k3)^b;
  return seed;

}

__device__ unsigned rng::LNG (unsigned seed, unsigned a, unsigned b) {

  return seed = (a*seed + b);

}

__device__ unsigned rng::hybrid() /*(unsigned s1, unsigned s2, unsigned s3, unsigned s4) */{

  return (2.3283064365387e-10*(TauStep(m_s1, 13, 19, 12, 4294967294UL) ^
  (TauStep(m_s2, 2, 25, 4, 4294967288UL) )^
  (TauStep(m_s3, 3, 11, 17, 4294967280UL))
  ^(LNG(m_s4, 1664525, 1013904223UL))));

}

/*__global__ void rng::combined_generator(unsigned * r) {

  int i = threadIdx.x + blockDim.x * blockIdx.x;

  r[i] = hybrid(m_s1, m_s2, m_s3, m_s4);

}
*/
