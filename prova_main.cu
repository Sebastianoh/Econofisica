#include "hip/hip_runtime.h"
// ############## MAIN PER FARE I TEST ############

#include <iostream>
#include <cmath>
#include <cstdlib>
#include "rng.h"
#include "rng.cu"
#include "struct.h"
#include "funzioni.h"
#include "montecarlo.h"
#include "montecarlo.cu"
#include "path.h"
#include "path.cu"

using namespace std;
//le global non posso definirle nelle classi

__global__ void test_rng(rng R, path P, double * array_prova) {

    rng  thread_rng  = R;
    path thread_path = P;
    double dummy;

    int i = threadIdx.x + blockDim.x*blockIdx.x;

      for (size_t j = 0; j < 10; j++) {
        dummy = thread_path.eulero(thread_rng.Get_gauss());
      }

    array_prova[i] = dummy;
};

int main() {

  // int N = 150;

  std::cout << "" << '\n';
  std::cout << "##########################################" << '\n';
  std::cout << "" << '\n';

  unsigned s1,s2,s3,s4;
  double * array = new double [100];
  double * dev_array = new double [100];

  input_market_data market;
  input_option_data option;
  input_mc_data     data_montecarlo;

  set_struct_values(market, option, data_montecarlo);
  random_seed_generator(s1,s2,s3,s4);

  rng random_number_generator(s1,s2,s3,s4);

  path path_creator(option, market);

  hipMalloc( (void **)&dev_array, 100*sizeof(double));

  test_rng<<<2,20>>>(random_number_generator, path_creator, dev_array);

  hipMemcpy(array, dev_array, 100*sizeof(double), hipMemcpyDeviceToHost);

  hipFree(dev_array);

    for (size_t i = 0; i < 10; i++) {
      std::cout << "array:" << array[i] << '\n';
    }

return 0;
}
