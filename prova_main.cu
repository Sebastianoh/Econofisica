#include "hip/hip_runtime.h"
// ############## MAIN PER FARE I TEST ############

#include <iostream>
#include <cmath>
#include <cstdlib>
#include "rng.h"
#include "rng.cu"
#include "struct.h"
#include "funzioni.h"
#include "processo_stocastico.h"
#include "processo_stocastico.cu"


using namespace std;
//le global non posso definirle nelle classi

__global__ void test_rng( double * array_prova) {

    // processo_stocastico host_pricer(10, 50, 12);

    // int i = threadIdx.x + blockDim.x*blockIdx.x;

    // processo_stocastico gpu_pricer(100, 1, 100);
    // uso -> perché gli passo una classe by pointer
    // array_prova[i] = host_pricer.Get_new_price();

};

int main() {

  // int N = 150;

  unsigned s1 = 0;
  unsigned s2 = 0;
  unsigned s3 = 0;
  unsigned s4 = 0;

  input_market_data market;
  input_option_data option;

  random_seed_generator(s1,s2,s3,s4);
  set_struct_values(market, option);

  rng random_number_generator(s1,s2,s3,s4);
  //struct option, struct data
  processo_stocastico host_pricer(option, market);

  for (size_t i = 0; i < 10; i++) {
    std::cout << "prova: " << host_pricer.Get_gauss() << '\n';
  }


return 0;
}
