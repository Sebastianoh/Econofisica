#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <cstdlib>
#include "funzioni.h"
#include "rng.h"
#include "rng.cu"
#include "processo_stocastico.h"
#include "processo_stocastico.cu"

using namespace std;
//le global non posso definirle nelle classi

__global__ void test_rng( double * array_prova) {

    processo_stocastico host_pricer(10, 50, 12); 
    
    int i = threadIdx.x + blockDim.x*blockIdx.x;

    // processo_stocastico gpu_pricer(100, 1, 100);
    // uso -> perché gli passo una classe by pointer
    array_prova[i] = host_pricer.Get_new_price();

};

int main() {

  int N = 150;

  unsigned s1 = 0;
  unsigned s2 = 0;
  unsigned s3 = 0;
  unsigned s4 = 0;

  random_seed_generator(s1,s2,s3,s4);

  // rng random_number_generator(s1,s2,s3,s4);
  processo_stocastico host_pricer(10, 50, 12);



// std::cout << "1:  " << host_pricer.Get_gauss()     << '\n';
// std::cout << "2:  " << host_pricer.Get_uniform()   << '\n';
// std::cout << "3:  " << host_pricer.Get_new_price() << '\n';


  double * prezzi     = new double [N];
  double * dev_prezzi = new double [N];

  //storage su cui copiare host pricer

  // processo_stocastico *dev_pricer;

  // hipMalloc( (void **)&dev_pricer, sizeof(processo_stocastico) );
  hipMalloc( (void **)&dev_prezzi, N*sizeof(double) );

  // copio:
  // hipMemcpy(&dev_pricer, &host_pricer, sizeof(processo_stocastico), hipMemcpyHostToDevice);

  //prova senza troppe pretese,
  // still non funziona
  test_rng<<<2,2>>>(dev_prezzi);

  hipMemcpy(prezzi, dev_prezzi, N*sizeof(double), hipMemcpyDeviceToHost);
  // hipFree(dev_pricer);

for (size_t i = 0; i < 10; i++) {
  std::cout << "prezzi:  " << prezzi[i] <<'\n';
  /* code */
}



  hipFree(dev_prezzi);

return 0;
}
