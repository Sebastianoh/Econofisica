#include "hip/hip_runtime.h"
// ############## MAIN PER FARE I TEST ############

#include <iostream>
#include <cmath>
#include <cstdlib>
#include "rng.h"
#include "rng.cu"
#include "struct.h"
#include "funzioni.h"
#include "montecarlo.h"
#include "montecarlo.cu"
#include "path.h"
#include "path.cu"

using namespace std;
//le global non posso definirle nelle classi


// __global__ void test_rng( double * array_prova) {

    // path host_pricer(10, 50, 12);

    // int i = threadIdx.x + blockDim.x*blockIdx.x;

    // path gpu_pricer(100, 1, 100);
    // uso -> perché gli passo una classe by pointer
    // array_prova[i] = host_pricer.Get_new_price();

// };

int main() {

  // int N = 150;

  std::cout << "" << '\n';
  std::cout << "##########################################" << '\n';
  std::cout << "" << '\n';

  unsigned s1,s2,s3,s4;

  input_market_data market;
  input_option_data option;
  input_mc_data     data_montecarlo;

  set_struct_values(market, option, data_montecarlo);
  random_seed_generator(s1,s2,s3,s4);

  rng random_number_generator(s1,s2,s3,s4);

  path path_creator(option, market);

  // ################ CREO UN PATH

  double final_price_test[data_montecarlo.N_tb][option.num_intervals];

    for (size_t j = 0; j < data_montecarlo.N_tb; j++) {


      for (size_t i = 0; i < option.num_intervals; i++) {

        final_price_test[j][i] = path_creator.eulero(random_number_generator.Get_gauss());
        std::cout << "final_price_test:   " << final_price_test[j][i] << '\n';

      }
    }
  std::cout << "" << '\n';
  std::cout << "##########################################" << '\n';
  std::cout << "" << '\n';

  // montecarlo          pricer_mc(data_montecarlo);

  //struct option, struct data



return 0;
}
