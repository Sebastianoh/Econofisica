#include "hip/hip_runtime.h"
// ############## MAIN PER FARE I TEST ############

#include <iostream>
#include <cmath>
#include <cstdlib>
#include "rng.h"
#include "rng.cu"
#include "struct.h"
#include "funzioni.h"
#include "path.h"

using namespace std;
//le global non posso definirle nelle classi

__device__ __host__ void pricer(double& media_prezzi, double& media_payoff, dati_anagrafici_opzione anagraf_data, input_option_data option, path thread_path, rng thread_rng, input_mc_data mc_data ) { //ipoteticamente posso passare il n_sim

  statistica stata(mc_data);
  payoff pay_off(anagraf_data, market_data, option_data);

  double * prezzi_finali = new double[mc_data.N_simulazioni];
  double * payoffs       = new double[mc_data.N_simulazioni];

    for (size_t i = 0; i < mc_data.N_simulazioni; i++) {

      double dummy = 0;

        for (size_t j = 0; j < option.num_intervals; j++) {

          dummy = thread_path.eulero(thread_rng.Get_gauss());

        }

      prezzi_finali[i] = dummy;
      payoffs[i] = pay_off.payoff_call(prezzi_finali[i]);

    }

  media_prezzi = stata.media(prezzi_finali);
  media_payoff = stata.media(payoffs);

// #### QUA IMPLEMENTO PAYOFF CORRIDOR ###


}


/*__global__ void simulator( input_mc_data mc_data, input_option_data option_data, input_market_data market_data) {
  int i = threadIdx.x + blockDim.x*blockIdx.x;

    rng thread_rng(1+i, 2345+i, 1234+i, 16545 +i);
  // dovrei passare tutte le classi e copiarle col copy constructor?
    path thread_path(option_data, market_data);



    // array_prova[i] = pricer(thread_path, thread_rng);
};
*/
int main() {

  // int N = 150;

  std::cout << "" << '\n';
  std::cout << "##########################################" << '\n';
  std::cout << "" << '\n';

  unsigned s1,s2,s3,s4;

  input_market_data market;
  input_option_data option;
  input_mc_data     data_montecarlo;

  set_struct_values(market, option, data_montecarlo);

  path path_creator(option, market);

  // hipMalloc( (void **)&dev_array, 100*sizeof(double));

  // test_rng<<<2,20>>>(random_number_generator, path_creator, dev_array);

  // hipMemcpy(array, dev_array, 100*sizeof(double), hipMemcpyDeviceToHost);

  // hipFree(dev_array);
